#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <cmath>
#include <random>

//using namespace std::chrono;

//void reportTime(const char* msg, steady_clock::duration span) {
//    auto ms = duration_cast<milliseconds>(span);
//    std::cout << msg << " - levou - " <<  ms.count() << " milisegundos" << std::endl;
//}

 // CRIE UMA FUNCTOR PARA CALCULAR A SQUARE

struct square 
{
   // const float a;
   // square(float _a): a(_a) {}
    __host__ __device__ // roda tanto na GPU quanto na CPU
    int operator()(int& x) {
        return x * x;
    }
};


 // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<int> vec) {
    float result;
    // ... add Thrust calls
    // AQUI VAO AS CHAMADAS THRUST
    thrust::device_vector<int> squared(vec.size()); 
    
    // chamar square
    thrust::transform(vec.begin(), vec.end(), squared.begin(), square());
    
    // fazer um reduce de soma
    result = sqrt(thrust::reduce(squared.begin(), squared.end(), (int)0, thrust::plus<int>()));

    return result;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    //steady_clock::time_point ts, te;

    // Faça um  vector em thrust 
    thrust::host_vector<int> d(n);
    


    // inicilize o  vector
    //ts = steady_clock::now();

    std::generate(d.begin(),d.end(), std::rand);

    thrust::device_vector<int> d_d = d;


    //te = steady_clock::now();
    //reportTime("Inicializacao", te - ts);

    // Calcule a magnitude do vetor
    //ts = steady_clock::now();
    float len = magnitude(d_d);
    //te = steady_clock::now();
    //reportTime("Tempo para calculo", te - ts);


    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
}
