#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <cmath>
#include <random>


// Carregar os dados do arquivo de entrada na memória da GPU
thrust::device_vector<int> start_times(N);
thrust::device_vector<int> end_times(N);
thrust::device_vector<int> categories(N);

// Ler os dados do arquivo de entrada
// ...
int M = 10;
int N = 10;

// Criar a matriz de programação dinâmica
thrust::device_vector<int> dp((N+1) * (M+1), 0);

// Inicializar a primeira linha da matriz com zeros
thrust::fill(dp.begin(), dp.begin() + M + 1, 0);

// Preencher a matriz com as soluções para subproblemas menores
for(int i = 1; i <= N; i++) {
    for (int j = 1; j <= M; j++) {
    // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
    int max_count = 0;
    for (int k = 0; k < i; k++) {
        if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(M+1)) + j-1] + 1 <= L[j-1]) {
        max_count = max(max_count, dp[(k*(M+1)) + j-1] + 1);
        } 
        else {
        max_count = max(max_count, dp[(k*(M+1)) + j]);
        }
    }
    dp[(i*(M+1)) + j] = max_count;
    }
}

// Encontrar o número máximo de filmes que podem ser assistidos
int max_count = 0;

for (int j = 1; j <= M; j++) {
    max_count = max(max_count, dp[(N*(M+1)) + j]);
}

// Escrever o resultado no arquivo de saída
// ...

